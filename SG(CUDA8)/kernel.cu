#include "hip/hip_runtime.h"


#include <stdio.h>
#include <random>


__global__ void Evolve(bool* field, float* scores, double b, int size, bool* next_field) {
	int row = threadIdx.y;
	int col = threadIdx.x;

	// Score
	if (col < size && row < size) {
		float score = 0;

		for (int i = -1; i <= 1; i++) //Row
		{
			for (int j = -1; j <= 1; j++) //Col
			{
				int memberIndex = (col + i + size) % size + size * ((row + j + size) % size);

				if (field[memberIndex] == 1)
					score++;
			}
		}

		if (!field[row*size + col])
			scores[row*size + col] = score * b;
		else 
			scores[row*size + col] = score;
	}

	__syncthreads();

	// Strategy
	int bestStrategyIndex = row*size + col;

	for (int i = -1; i <= 1; i++) //Row
	{
		for (int j = -1; j <= 1; j++) //Col
		{				
			int memberIndex = (col + i + size) % size + size * ((row + j + size) % size);

			if (scores[bestStrategyIndex] < scores[memberIndex]) 
			{
				bestStrategyIndex = memberIndex;
			}
		}
	}

	next_field[row*size + col] = field[bestStrategyIndex];
}



void InitField(bool* field, size_t size) {
	for (size_t i = 0; i < size*size; i++) {
		field[i] = rand() % 2;
	}
}

void InitScores(float* scores, size_t size) {
	for (size_t i = 0; i < size*size; i++) {
		scores[i] = 0;
	}
}

void PrintField(bool* field, int size) {
	for (int i = -1; i < size; i++) {
		for (size_t j = 0; j < size; j++)
		{
			if (i == -1) 
				printf("_");
			else
				printf("%s", field[i*size + j]? " " : "#");
		}
		printf("\n");
	}
}

void PrintScores(float* scores, size_t size) {
	for (size_t i = -1; i < size; i++) {
		for (size_t j = 0; j < size; j++)
		{
			if (i == -1) 
				printf("_");
			else
				printf("%.1f ", scores[i*size + j]);
		}
		printf("\n");
	}
}

int main()
{
	bool* field;
	size_t size = 10;
	double b = 1.81;

	dim3 block(size, size);

	bool *d_field, *d_next_field;
	float *d_scores;

	field = (bool*)malloc(sizeof(bool)*size*size);
	
	// GPU Memory
	hipMalloc((void**)&d_field, sizeof(bool)*size*size);
	hipMalloc((void**)&d_scores, sizeof(float)*size*size);
	hipMalloc((void**)&d_next_field, sizeof(bool)*size*size);

	InitField(field, size);
	PrintField(field, size);

	for (int i = 0; i < 10; i++) {
		// Init scores with zeros in GPU Memory
		hipMemcpy(d_field, field, size*size, hipMemcpyKind::hipMemcpyHostToDevice);

		hipMemset(d_scores, 0, size*size);		
	
		//Scores<<<1, block>>>(d_field, size, b, d_scores);
		//Strategy<<<1, block>>>(d_field, d_scores, size, d_next_field);		

		Evolve<<<1, block>>>(d_field, d_scores, b, size, d_next_field);

		hipMemcpy(field, d_next_field, size*size, hipMemcpyKind::hipMemcpyDeviceToHost);

		PrintField(field, size);
	}

	hipFree(d_field);
	hipFree(d_next_field);
	hipFree(d_scores);

	getchar();

    return 0;
}
